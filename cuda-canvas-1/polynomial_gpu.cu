#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>


void polynomial_expansion (float* poly, int degree,
			   int n, float* array) {
  //TODO: Write code to use the GPU here!
  //code should write the output back to array
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if( index < n )
  {
    float out = 0.0;
    float xtothepowerof = 1.0;
    for ( int i = 0; i < degree+1; ++i)
    {
      out += xtothepowerof * poly[i];
      xtothepowerof *= array[index];
    }
    array[index] = out;
  }
}


int main (int argc, char* argv[]) {
  //TODO: add usage
  
  if (argc < 3) {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]); //TODO: atoi is an unsafe function
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

  float *dev_array, *dev_poly;
  
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();
  
  //for (int iter = 0; iter<nbiter; ++iter)
    //polynomial_expansion (poly, degree, n, array);
    
  hipMallocManaged(&dev_array, n*sizeof(float));
  hipMallocManaged(&dev_poly, (degree+1)*sizeof(float));

  hipMemcpy(dev_array, array, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_poly, poly, (degree+1)*sizeof(float), hipMemcpyHostToDevice);

  polynomial_expansion<<<(n+255)/256, 256>>>(dev_poly, degree, n, dev_array);
  hipMemcpy(array, dev_array, n*sizeof(float), hipMemcpyDeviceToHost);
  hipFree(dev_array);
  hipFree(dev_poly);
  hipDeviceSynchronize();

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  {
    bool correct = true;
    int ind;
    for (int i=0; i< n; ++i) {
      if (fabs(array[i]-(degree+1))>0.01) {
        correct = false;
	ind = i;
      }
    }
    if (!correct)
      std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }
  

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  delete[] array;
  delete[] poly;

  return 0;
}
